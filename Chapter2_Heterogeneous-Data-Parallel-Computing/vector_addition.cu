// compilation: nvcc vector_addition.cu -o vector_addition
// usage: ./vector_addition

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N 100000000

void vecAddRegular(float* A_h, float* B_h, float* C_h, int n) {
    for (int i = 0; i < n; i++) {
        C_h[i] = A_h[i] + B_h[i];
    }
}

void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
    int size = sizeof(float) * n; 
    // on-device versions
    float *A_d, *B_d, *C_d; 
    // allocate memory on device
    hipMalloc((void**)&A_d, size); 
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);
    // copy data from host to device
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);



    // copy result back to host
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    // free on-device memory
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main() {

    // Just do regular C here
    float *A, *B, *C;

    // allocate memory -- note regular C
    // malloc only takes size as input
    int size = sizeof(float) * N;
    A = (float*)malloc(size); 
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // initialize
    for (int i = 0; i < N; i++) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    // run regular addition
    clock_t start = clock();
    vecAddRegular(A, B, C, N); 
    // Record the end time
    clock_t end = clock();
    // Calculate the elapsed time in seconds
    double elapsed_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Regular vector addition: %.3f seconds\n", elapsed_time);


    // now do the parallel / on-device version


}


